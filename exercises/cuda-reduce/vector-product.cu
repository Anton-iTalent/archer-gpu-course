/*
 * Exercise.
 * Compute the vector product of two double arrays a and b.
 *
 * Training material developed by James Perry and Alan Gray
 * Copyright EPCC, The University of Edinburgh, 2010 
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime_api.h>

/* Utility function to check for and report CUDA errors */
void checkCUDAError(const char*);

/* The number of integer elements in the array */
#define ARRAY_SIZE 2048

/*
 * The number of CUDA blocks and threads per block to use.
 */

#define THREADS_PER_BLOCK 256

/* The kernel is here: */

__global__ void vector_product(double * a, double * b, double * result) {

/* PART 2: implemnet the kernel to compute the correct result */

}

/* Main routine */

int main(int argc, char *argv[]) {

  size_t sz = ARRAY_SIZE*sizeof(double);

  double * h_a = NULL;
  double * h_b = NULL;
  double * d_a = NULL;
  double * d_b = NULL;

  /* Print device details */

  int nDevice = 0;

  hipGetDeviceCount(&nDevice);
  if (nDevice == 0) {
    printf("No devices detected. Make sure you run in the queue system.\n");
    exit(-1);
  }
  else {
    int deviceNum;
    hipDeviceProp_t prop;
    hipGetDevice(&deviceNum);	
    hipGetDeviceProperties(&prop, deviceNum);
    printf("  Device name: %s\n", prop.name);
  }   

  /*
   * allocate memory on host
   */

  h_a = (double *) malloc(sz);
  h_b = (double *) malloc(sz);

  /*
   * allocate memory on device
   */

  hipMalloc((void **) &d_a, sz);
  hipMalloc((void **) &d_b, sz);

  /* initialise host arrays */

  for (int i = 0; i < ARRAY_SIZE; i++) {
    h_a[i] = 1.0;
    h_b[i] = 2.0;
  }

  hipMemcpy(d_a, h_a, sz, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, sz, hipMemcpyHostToDevice);

  /* PART 1: allocate and initialise device result */
  
  /* run the kernel on the GPU */

  {
    dim3 nthreads = THREADS_PER_BLOCK;	
    dim3 nblocks  = 1 + (ARRAY_SIZE - 1)/THREADS_PER_BLOCK;
    /* PART 2: Kernel */
  }   

  /* wait for all threads to complete and check for errors */

  hipDeviceSynchronize();
  checkCUDAError("kernel invocation");

  /* Result */

  printf("Results:\n");

  {
    double h_sum = 0.0;
    double h_sum_expect = 0.0;

    /* PART 1: Copy result back to host here ... */

    for (int i = 0; i < ARRAY_SIZE; i++) {
      h_sum_expect += h_a[i]*h_b[i];
    }
    printf("Host result:   %14.7e\n", h_sum_expect);
    printf("Kernel result: %14.7e\n", h_sum);
  }
  printf("\n");

  /* free device buffers */

  hipFree(d_b);
  hipFree(d_a);

  /* free host buffers */

  free(h_a);
  free(h_b);

  return 0;
}

/* Utility function to check for and report CUDA errors */

void checkCUDAError(const char * msg) {

  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}
